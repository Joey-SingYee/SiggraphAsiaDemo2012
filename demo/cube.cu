/*
Siggraph Asia 2012 Demo

Mass vector implementation.

Laurence Emms
*/

#include <iostream>
#include <iomanip>
#include <vector>
#include <fstream>
#include <iterator>

#include <GL/glew.h>
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

// GLM
#define GLM_FORCE_CUDA
#include <glm/glm.hpp>
#include <glm/gtc/type_ptr.hpp>
#include <glm/gtc/matrix_transform.hpp>

#include "mass.h"
#include "spring.h"
#include "creator.h"
#include "cube.h"

SigAsiaDemo::Cube::Cube(
	size_t x, // multiple of 2
	size_t y,
	size_t z,
	float spacing,
	float mass,
	float radius) :
		_start(0),
		_end(0),
		_half_x(static_cast<int>(x/2)),
		_half_y(static_cast<int>(y/2)),
		_half_z(static_cast<int>(z/2)),
		_spacing(spacing),
		_mass(mass),
		_radius(radius)
{}

SigAsiaDemo::Cube::~Cube()
{} 

void SigAsiaDemo::Cube::create(
	float x,
	float y,
	float z,
	MassList &masses,
	SpringList &springs)
{
	_start = masses.size();
	std::cout << "Starting at index " << _start << "." << std::endl;

	int side = _half_x*2+1;
	int plane = side*side;
	//int cube = plane*side;

	// add points
	for (int i = -_half_x; i <= _half_x; ++i) {
		for (int j = -_half_y; j <= _half_y; ++j) {
			for (int k = -_half_z; k <= _half_z; ++k) {
				masses.push(SigAsiaDemo::Mass(
					_mass,
					static_cast<float>(i)*_spacing + x,
					static_cast<float>(j)*_spacing + y,
					static_cast<float>(k)*_spacing + z,
					0.0, 0.0, 0.0,
					0,
					_radius));
			}
		}
	}
	_end = masses.size();
	std::cout << "Ending at index " << _end << "." << std::endl;

	// add structural springs
	for (int i = -_half_x; i <= _half_x; ++i) {
		for (int j = -_half_y; j <= _half_y; ++j) {
			for (int k = -_half_z; k <= _half_z; ++k) {
				int ind_i = i + _half_x;
				int ind_j = j + _half_y;
				int ind_k = k + _half_z;

				int index = _start + ind_i + ind_j*side + ind_k*plane;
				// add springs to neighbors
				int right = -1;
				int down = -1;
				int back = -1;

				// compute indices
				if (i+1 <= _half_x)
					right = _start + (ind_i+1) + ind_j*side + ind_k*plane;
				if (j+1 <= _half_y)
					down = _start + ind_i + (ind_j+1)*side + ind_k*plane;
				if (k+1 <= _half_z)
					back = _start + ind_i + ind_j*side + (ind_k+1)*plane;

				// add springs
				if (right >= 0) {
					springs.push(SigAsiaDemo::Spring(
						masses,
						index,
						right));
				}

				if (down >= 0) {
					springs.push(SigAsiaDemo::Spring(
						masses,
						index,
						down));
				}

				if (back >= 0) {
					springs.push(SigAsiaDemo::Spring(
						masses,
						index,
						back));
				}
			}
		}
	}

	// add bending springs
	for (int i = -_half_x; i <= _half_x; ++i) {
		for (int j = -_half_y; j <= _half_y; ++j) {
			for (int k = -_half_z; k <= _half_z; ++k) {
				int ind_i = i + _half_x;
				int ind_j = j + _half_y;
				int ind_k = k + _half_z;

				int index = _start + ind_i + ind_j*side + ind_k*plane;
				// add springs to neighbors
				int right = -1;
				int down = -1;
				int back = -1;

				// compute indices
				if (i+2 <= _half_x)
					right = _start + (ind_i+2) + ind_j*side + ind_k*plane;
				if (j+2 <= _half_y)
					down = _start + ind_i + (ind_j+2)*side + ind_k*plane;
				if (k+2 <= _half_z)
					back = _start + ind_i + ind_j*side + (ind_k+2)*plane;

				// add springs
				if (right >= 0) {
					springs.push(SigAsiaDemo::Spring(
						masses,
						index,
						right));
				}

				if (down >= 0) {
					springs.push(SigAsiaDemo::Spring(
						masses,
						index,
						down));
				}

				if (back >= 0) {
					springs.push(SigAsiaDemo::Spring(
						masses,
						index,
						back));
				}
			}
		}
	}

	// add shear springs
	for (int i = -_half_x; i <= _half_x; ++i) {
		for (int j = -_half_y; j <= _half_y; ++j) {
			for (int k = -_half_z; k <= _half_z; ++k) {
				int ind_i = i + _half_x;
				int ind_j = j + _half_y;
				int ind_k = k + _half_z;

				// add springs to neighbors

				// front plane indices
				int index = _start + ind_i + ind_j*side + ind_k*plane;
				int right = -1;
				int down = -1;
				int right_down = -1;

				// back plane indices
				int back = -1;
				int back_right = -1;
				int back_down = -1;
				int back_right_down = -1;

				// compute indices
				if (i+1 <= _half_x) {
					right = _start +
					(ind_i+1) +
					ind_j*side +
					ind_k*plane;
				}
				if (j+1 <= _half_y) {
					down =
						_start +
						ind_i +
						(ind_j+1)*side +
						ind_k*plane;
				}
				if (i+1 <= _half_x && j+1 <= _half_y) {
					right_down =
						_start +
						(ind_i+1) +
						(ind_j+1)*side +
						ind_k*plane;
				}

				if (k+1 <= _half_z) {
					back = _start + ind_i + ind_j*side + (ind_k+1)*plane;
					if (i+1 <= _half_x) {
						back_right =
							_start +
							(ind_i+1) +
							ind_j*side +
							(ind_k+1)*plane;
					}
					if (j+1 <= _half_y) {
						back_down =
							_start +
							ind_i +
							(ind_j+1)*side +
							(ind_k+1)*plane;
					}
					if (i+1 <= _half_x && j+1 <= _half_y) {
						back_right_down = 
							_start +
							(ind_i+1) +
							(ind_j+1)*side +
							(ind_k+1)*plane;
					}

				}

				// add planar springs
				// front plane
				if (right >= 0 && down >= 0) {
					springs.push(SigAsiaDemo::Spring(
						masses,
						index,
						right_down));
					springs.push(SigAsiaDemo::Spring(
						masses,
						right,
						down));
				}

				// left plane
				if (back >= 0 && down >= 0) {
					springs.push(SigAsiaDemo::Spring(
						masses,
						index,
						back_down));
					springs.push(SigAsiaDemo::Spring(
						masses,
						back,
						down));
				}

				// top plane
				if (back >= 0 && right >= 0) {
					springs.push(SigAsiaDemo::Spring(
						masses,
						index,
						back_right));
					springs.push(SigAsiaDemo::Spring(
						masses,
						back,
						right));
				}

				if (back_right_down >= 0) {
					// back plane
					springs.push(SigAsiaDemo::Spring(
						masses,
						back,
						back_right_down));
					springs.push(SigAsiaDemo::Spring(
						masses,
						back_right,
						back_down));

					// right plane
					springs.push(SigAsiaDemo::Spring(
						masses,
						right,
						back_right_down));
					springs.push(SigAsiaDemo::Spring(
						masses,
						back_right,
						right_down));

					// bottom plane
					springs.push(SigAsiaDemo::Spring(
						masses,
						down,
						back_right_down));
					springs.push(SigAsiaDemo::Spring(
						masses,
						back_down,
						right_down));
				}
			}
		}
	}
}
