#include "hip/hip_runtime.h"
/*
Siggraph Asia 2012 Demo

Spring vector implementation.

Laurence Emms
*/

#include <iostream>
#include <iomanip>
#include <vector>
#include <cmath>
#include "mass.h"
#include "spring.h"

SigAsiaDemo::Spring::Spring(
	MassList &masses,
	unsigned int mass0,
	unsigned int mass1,
	float ks,
	float kd) :
		_mass0(mass0),
		_mass1(mass1),
		_ks(ks),
		_kd(kd),
		_l0(0.0),
		_fx0(0.0),
		_fy0(0.0),
		_fz0(0.0),
		_fx1(0.0),
		_fy1(0.0),
		_fz1(0.0)
{
	// compute l0
	Mass *m0 = masses.getMass(_mass0);
	Mass *m1 = masses.getMass(_mass1);
	if (!m0) {
		std::cerr << "Spring pointing to null mass 0" << std::endl;
		std::terminate();
	}
	if (!m1) {
		std::cerr << "Spring pointing to null mass 1" << std::endl;
		std::terminate();
	}
	float dx = m0->_x - m1->_x;
	float dy = m0->_y - m1->_y;
	float dz = m0->_z - m1->_z;
	_l0 = sqrt(dx*dx + dy*dy + dz*dz);
}

SigAsiaDemo::SpringList::SpringList() :
	_computing(false),
	_changed(false),
	_device_springs(0),
	_device_mass_spring_counts(0),
	_device_mass_spring_indices(0)
{
}

SigAsiaDemo::SpringList::~SpringList()
{
	if (_computing) {
		std::cout << "Warning: Still computing!" << std::endl;
	}
	if (_device_springs) {
		std::cout << "Free springs." << std::endl;
		hipFree(_device_springs);
		_device_springs = 0;
	}

	if (_device_mass_spring_counts) {
		std::cout << "Free counts." << std::endl;
		hipFree(_device_mass_spring_counts);
		_device_mass_spring_counts = 0;
	}

	if (_device_mass_spring_indices) {
		std::cout << "Free indices." << std::endl;
		hipFree(_device_mass_spring_indices);
		_device_mass_spring_indices = 0;
	}
}

bool SigAsiaDemo::SpringList::push(Spring spring)
{
	// enforce that no springs can be added
	// if the buffer is uploaded to the GPU
	if (!_computing) {
		_springs.push_back(spring);
		_changed = true;
		return true;
	}
	return false;
}

bool SigAsiaDemo::SpringList::empty() const
{
	return _springs.empty();
}

size_t SigAsiaDemo::SpringList::size() const
{
	return _springs.size();
}

// Note: this must be called before updating the mass list
void SigAsiaDemo::SpringList::upload(MassList &masses)
{
	if (_computing) {
		// do nothing if computing
		return;
	}

	// if masses have been changed or springs have been changed, the mapping
	// between masses and springs will need to be recomputed
	
	// NOTE: we assume that the mass list has not changed, otherwise the spring
	// list is invalid and needs to be changed as well
	if (_changed) {
		std::cout << "Update mass -> spring mapping." << std::endl;
		// NOTE: hopefully this won't change too often, otherwise a GPU 
		// solution will have to be developed for updating this mapping

		// clear counts and indices
		_mass_spring_counts.clear();
		_mass_spring_indices.clear();
		size_t count = 0;
		size_t counts_index = 1;
		size_t indices_index = 0;
		// contains the number of spring indices for each mass
		// the last element of this array is the total indices
		// array size
		_mass_spring_counts.resize(masses.size() + 1);
		_mass_spring_indices.resize(_springs.size() * 2);
		_mass_spring_counts[0] = 0;

		for (size_t i = 0; i < masses.size(); ++i) {
			const Mass *m = masses.getMass(i);
			if (!m) {
				std::cout << "Error: Failed to get mass " << i << std::endl;
				std::terminate();
			}
			for (size_t j = 0; j < _springs.size(); ++j) {
				if (_springs[j]._mass0 == i || _springs[j]._mass1 == i) {
					if (_springs[j]._mass0 == i && _springs[j]._mass1 == i) {
						std::cerr << \
						"Error: Both spring indices point to the same mass." \
						<< std::endl;
						std::terminate();
					}

					// increment counts for this mass
					std::cout << "Mass has spring index: " << j << std::endl;
					if (indices_index >= _mass_spring_indices.size()) {
						std::cout << "Error: indices_index exceeds expected \
indices size." << std::endl;
						std::terminate();
					}
					_mass_spring_indices[indices_index] = j;
					indices_index++;
					count++;
				}
			}
			if (counts_index >= _mass_spring_counts.size()) {
				std::cout << "Error: counts_index exceeds expected \
counts size." << std::endl;
				std::terminate();
			}
			_mass_spring_counts[counts_index] = count;
			counts_index++;
		}
		
		// TODO: remove
		std::cout << "counts: " << _mass_spring_counts.size() << std::endl;
		std::cout << "indices: " << _mass_spring_indices.size() << std::endl;
		std::cout << "mass spring indices:" << std::endl;
		for (size_t i = 0; i < masses.size(); ++i) {
			std::cout << _mass_spring_counts[i] << " -> " \
			<< _mass_spring_counts[i+1] << ":" << std::endl;

			for (size_t j = _mass_spring_counts[i];
				j < _mass_spring_counts[i+1]; ++j) {
				std::cout << _mass_spring_indices[j] << ", ";
			}

			std::cout << std::endl;
		}

		std::cout << "Upload mass -> spring mapping." << std::endl;
		if (_device_mass_spring_counts) {
			std::cout << "Free counts." << std::endl;
			hipFree(_device_mass_spring_counts);
			_device_mass_spring_counts = 0;
		}
		if (_device_mass_spring_indices) {
			std::cout << "Free indices." << std::endl;
			hipFree(_device_mass_spring_indices);
			_device_mass_spring_indices = 0;
		}

		// allocate GPU buffers
		std::cout << std::fixed << std::setprecision(8) \
		<< "Allocate GPU counts buffer of size " \
		<< _mass_spring_counts.size()*sizeof(unsigned int)/1073741824.0 \
		<< " GB." << std::endl;
		hipError_t result = hipMalloc(
			(void**)&_device_mass_spring_counts,
			_mass_spring_counts.size()*sizeof(unsigned int));
		if (result != hipSuccess) {
			std::cout << "Error: CUDA failed to malloc memory." << std::endl;
			std::terminate();
		}

		std::cout << std::fixed << std::setprecision(8) \
		<< "Allocate GPU indices buffer of size " \
		<< _mass_spring_indices.size()*sizeof(unsigned int)/1073741824.0 \
		<< " GB." << std::endl;
		result = hipMalloc(
			(void**)&_device_mass_spring_indices,
			_mass_spring_indices.size()*sizeof(unsigned int));
		if (result != hipSuccess) {
			std::cout << "Error: CUDA failed to malloc memory." << std::endl;
			std::terminate();
		}

		// copy into GPU buffer
		std::cout << "Copy counts into GPU buffer." << std::endl;
		hipMemcpy(
			_device_mass_spring_counts,
			&_mass_spring_counts[0],
			_mass_spring_counts.size()*sizeof(unsigned int),
			hipMemcpyHostToDevice);

		std::cout << "Copy indices into GPU buffer." << std::endl;
		hipMemcpy(
			_device_mass_spring_indices,
			&_mass_spring_indices[0],
			_mass_spring_indices.size()*sizeof(unsigned int),
			hipMemcpyHostToDevice);

		std::cout << "Upload springs." << std::endl;
		_changed = false;
		if (_device_springs) {
			std::cout << "Free springs." << std::endl;
			hipFree(_device_springs);
			_device_springs = 0;
		}

		// allocate GPU buffer
		std::cout << std::fixed << std::setprecision(8) \
		<< "Allocate GPU buffer of size " << \
		_springs.size()*sizeof(Spring)/1073741824.0 \
		<< " GB." << std::endl;
		result = hipMalloc(
			(void**)&_device_springs,
			_springs.size()*sizeof(Spring));
		if (result != hipSuccess) {
			std::cout << "Error: CUDA failed to malloc memory." << std::endl;
			std::terminate();
		}

		// copy into GPU buffer
		std::cout << "Copy springs into GPU buffer." << std::endl;
		hipMemcpy(
			_device_springs,
			&_springs[0],
			_springs.size()*sizeof(Spring),
			hipMemcpyHostToDevice);
	}

	_computing = true;
}

void SigAsiaDemo::SpringList::download()
{
	if (_changed) {
		std::cerr << "Error: Spring list changed while \
data was being used in GPU computations." << std::endl;
		std::terminate();
	} else {
		std::cout << "Download springs." << std::endl;
		// copy into CPU buffer
		std::cout << "Copy springs into CPU buffer." << std::endl;
		hipMemcpy(
			&_springs[0],
			_device_springs,
			_springs.size()*sizeof(Spring),
			hipMemcpyDeviceToHost);
	}
	_computing = false;
}

SigAsiaDemo::Spring *SigAsiaDemo::SpringList::getSpring(size_t index)
{
	if (_springs.empty()) {
		std::cout << "Warning: getSpring called on \
empty spring list." << std::endl;
		return 0;
	}
	if (index >= _springs.size()) {
		std::cout << "Warning: getSpring called on index \
out of bounds." << std::endl;
		return 0;
	}

	return &_springs[index];
}

SigAsiaDemo::Spring *SigAsiaDemo::SpringList::getDeviceSprings()
{
	return _device_springs;
}

bool SigAsiaDemo::SpringList::getChanged() const
{
	return _changed;
}

__global__ void deviceComputeSpringForces(
	unsigned int springs_size,
	SigAsiaDemo::Spring *springs,
	unsigned int masses_size,
	SigAsiaDemo::Mass *masses)
{
	int tid = blockIdx.x;
	if (tid < springs_size) {
		// v is the vector from mass 1 to mass 0
		// we're operating on the temporary position
		float vx =
			masses[springs[tid]._mass0]._tx - masses[springs[tid]._mass1]._tx;
		float vy =
			masses[springs[tid]._mass0]._ty - masses[springs[tid]._mass1]._ty;
		float vz =
			masses[springs[tid]._mass0]._tz - masses[springs[tid]._mass1]._tz;
		// compute length of v
		float lv = sqrt(vx*vx + vy*vy + vz*vz);
		float rcp_lv = 1.0f;
		if (lv != 0.0f) {
			rcp_lv = 1.0f / lv;
		}
		// compute unit v
		float uvx = vx * rcp_lv;
		float uvy = vy * rcp_lv;
		float uvz = vz * rcp_lv;

		// project temporary velocity of mass 0 onto v
		float dot_tv0_v =
			masses[springs[tid]._mass0]._tvx * uvx +
			masses[springs[tid]._mass0]._tvy * uvy +
			masses[springs[tid]._mass0]._tvz * uvz;
		float tv0x = uvx * dot_tv0_v;
		float tv0y = uvy * dot_tv0_v;
		float tv0z = uvz * dot_tv0_v;
		// compute force for mass 0 to mass 1
		float extension = -springs[tid]._ks * (lv / springs[tid]._l0 - 1.0f);
		springs[tid]._fx0 = extension * uvx - springs[tid]._kd * tv0x;
		springs[tid]._fy0 = extension * uvy - springs[tid]._kd * tv0y;
		springs[tid]._fz0 = extension * uvz - springs[tid]._kd * tv0z;

		// project temporary velocity of mass 1 onto -v
		float dot_tv1_v =
			masses[springs[tid]._mass1]._tvx * uvx +
			masses[springs[tid]._mass1]._tvy * uvy +
			masses[springs[tid]._mass1]._tvz * uvz;
		float tv1x = -uvx * dot_tv1_v;
		float tv1y = -uvy * dot_tv1_v;
		float tv1z = -uvz * dot_tv1_v;
		// compute force for mass 1 to mass 0
		extension = springs[tid]._ks * (lv / springs[tid]._l0 - 1.0f);
		springs[tid]._fx1 = extension * uvx - springs[tid]._kd * tv1x;
		springs[tid]._fy1 = extension * uvy - springs[tid]._kd * tv1y;
		springs[tid]._fz1 = extension * uvz - springs[tid]._kd * tv1z;
	}
}

__global__ void deviceApplySpringForces(
	unsigned int springs_size,
	SigAsiaDemo::Spring *springs,
	unsigned int masses_size,
	SigAsiaDemo::Mass *masses,
	unsigned int *mass_spring_counts,
	unsigned int *mass_spring_indices)
{
	int tid = blockIdx.x;
	if (tid < masses_size) {
		for (unsigned int i = mass_spring_counts[tid];
			i < mass_spring_counts[tid+1];
			++i) {
			unsigned int s = mass_spring_indices[i];
			if (tid == springs[s]._mass0) {
				masses[tid]._fx += springs[s]._fx0;
				masses[tid]._fy += springs[s]._fy0;
				masses[tid]._fz += springs[s]._fz0;
			} else if (tid == springs[s]._mass1) {
				masses[tid]._fx += springs[s]._fx1;
				masses[tid]._fy += springs[s]._fy1;
				masses[tid]._fz += springs[s]._fz1;
			}
		}
	}
}

void SigAsiaDemo::SpringList::applySpringForces(MassList &masses)
{
	if (_computing && !_springs.empty() && !masses.empty()) {
		std::cout << "Compute spring forces (" << _springs.size() << ")." \
		<< std::endl;
		deviceComputeSpringForces<<<_springs.size(), 1>>>(
			_springs.size(),
			_device_springs,
			masses.size(),
			masses.getDeviceMasses());
		hipDeviceSynchronize();

		std::cout << "Accumulate mass forces (" << masses.size() << ")." \
		<< std::endl;
		deviceApplySpringForces<<<masses.size(), 1>>>(
			_springs.size(),
			_device_springs,
			masses.size(),
			masses.getDeviceMasses(),
			_device_mass_spring_counts,
			_device_mass_spring_indices);
		hipDeviceSynchronize();
	}
}
